﻿#include "known_primes.h"
#include "sieve.cuh"
#include "print.h"

#define GPU_ENABLED 1
#define PRINT_TO_FILE 1

constexpr size_t two_factor_sqrt(size_t n, size_t s = 0)
{
	constexpr auto LARGEST_INPUT = 1ull << 62;
	if (n > LARGEST_INPUT) return 1ull << 32;
	size_t i = 1ull << s;
	return i * i >= n ? i : two_factor_sqrt(n, s + 1);
}

int main()
{
	constexpr auto ARRAY_SIZE = 1ull << 32;
	constexpr auto FINAL_NUMBER = 1ull << 34;
	constexpr auto SIEVE_CALLS = FINAL_NUMBER / ARRAY_SIZE;

	auto knownPrimes = get_known_primes(two_factor_sqrt(ARRAY_SIZE));
	Array<bool> result;
	result.ptr = (bool*)malloc(ARRAY_SIZE);
	result.size = ARRAY_SIZE;
	Printer printer;
	for (auto offset = 0ull; offset < FINAL_NUMBER; offset += ARRAY_SIZE)
	{
		std::cout << "\nSIEVE CALL " << (offset / ARRAY_SIZE + 1ull) << " OUT OF " << SIEVE_CALLS << std::endl;
		sieve<GPU_ENABLED>(result, offset, knownPrimes);
		printer.print<PRINT_TO_FILE>(result, offset);
		if (offset == 0ull)
			update_known_primes(knownPrimes, result, two_factor_sqrt(FINAL_NUMBER));
	}
	printer.writeToFile();
	free(result.ptr);

	return 0;
}
